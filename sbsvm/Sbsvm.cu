#include "org_sbsvm_Sbsvm.h"

#include <iostream>

#include <hip/hip_runtime.h>

#include "cptr.h"
#include "example.h"

static jclass ByteBuffer;
static jmethodID allocateDirect;

extern "C" {
/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    initialize
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_initialize
  (JNIEnv *env, jobject o)
{
  ByteBuffer = env->FindClass("java/nio/ByteBuffer");
  allocateDirect = env->GetStaticMethodID(ByteBuffer, "allocateDirect", "(I)Ljava/nio/ByteBuffer;");
  cpu_pointer::initialize();
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    finalize
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_finalize
  (JNIEnv *env, jobject o)
{
  cpu_pointer::finalize();
}


/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    clear
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_clear
  (JNIEnv *env, jobject o)
{
  cpu_pointer::clear_cache();
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    run
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_run
  (JNIEnv *env, jobject o)
{
  cpu_pointer::run_handler();
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    loadModule
 * Signature: (Ljava/nio/ByteBuffer;)Ljava/nio/ByteBuffer;
 */
JNIEXPORT jobject JNICALL Java_org_sbsvm_Sbsvm_loadModule
  (JNIEnv *env, jobject o, jobject image)
{
  const void *pImage = env->GetDirectBufferAddress(image);
  hipModule_t module;
  hipModuleLoadData(&module, pImage);
  jobject ret = env->CallStaticObjectMethod(ByteBuffer, allocateDirect, (jint)sizeof(hipModule_t));
  hipModule_t *pModule = reinterpret_cast<hipModule_t*>(env->GetDirectBufferAddress(ret));
  *pModule = module;
  return ret;
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    getFunction
 * Signature: (Ljava/nio/ByteBuffer;Ljava/nio/ByteBuffer;)Ljava/nio/ByteBuffer;
 */
JNIEXPORT jobject JNICALL Java_org_sbsvm_Sbsvm_getFunction
  (JNIEnv *env, jobject o, jobject module, jobject name)
{
  hipModule_t *pModule = reinterpret_cast<hipModule_t*>(env->GetDirectBufferAddress(module));
  char *pName = reinterpret_cast<char*>(env->GetDirectBufferAddress(name));
  hipFunction_t func;
  std::cout << "0!!!!!!!!!!!!!!!!!" << std::endl;
  hipModuleGetFunction(&func, *pModule, pName);
  std::cout << "1!!!!!!!!!!!!!!!!!" << std::endl;
  jobject ret = env->CallStaticObjectMethod(ByteBuffer, allocateDirect, (jint)sizeof(hipFunction_t));
  hipFunction_t *pFunc = reinterpret_cast<hipFunction_t*>(env->GetDirectBufferAddress(ret));
  *pFunc = func;
  return ret;

}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    createStream
 * Signature: ()Ljava/nio/ByteBuffer;
 */
JNIEXPORT jobject JNICALL Java_org_sbsvm_Sbsvm_createStream
  (JNIEnv *env, jobject o)
{
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
  jobject ret = env->CallStaticObjectMethod(ByteBuffer, allocateDirect, (jint)sizeof(hipStream_t));
  hipStream_t *pStream = reinterpret_cast<hipStream_t*>(env->GetDirectBufferAddress(ret));
  *pStream = stream;
  return ret;
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    launchKernel
 * Signature: (Ljava/nio/ByteBuffer;JJJJJJJLjava/nio/ByteBuffer;)V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_launchKernel
  (JNIEnv *env, jobject o, jobject function, jlong gridDimX, jlong gridDimY, jlong gridDimZ, jlong blockDimX, jlong blockDimY, jlong blockDimZ, jlong sharedMemBytes, jobject stream)
{
  hipFunction_t *pFunc = reinterpret_cast<hipFunction_t*>(env->GetDirectBufferAddress(function));
  hipStream_t *pStream = reinterpret_cast<hipStream_t*>(env->GetDirectBufferAddress(stream));
  hipModuleLaunchKernel(*pFunc, gridDimX, gridDimY, gridDimZ, blockDimX, blockDimY, blockDimZ, sharedMemBytes, *pStream, nullptr, nullptr);
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    test
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_test
  (JNIEnv *env, jobject o)
{
  float (*A)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  float (*B)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  float (*C)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  dim3 block(32, 8);
  dim3 grid(MAT_SIZE/block.x+(MAT_SIZE%block.x!=0), MAT_SIZE/block.y+(MAT_SIZE%block.y!=0));
  gpu_client<<<grid, block>>>(A, B, C);
}

} // extern "C"
