#include "hip/hip_runtime.h"
#include "org_sbsvm_Sbsvm.h"

#include "cptr.h"

#define MAT_SIZE (1*1024ul)

__global__ void gpu_client(float (*A)[MAT_SIZE], float (*B)[MAT_SIZE], float (*C)[MAT_SIZE])
{
  const int j = blockIdx.x * blockDim.x + threadIdx.x;
  const int i = blockIdx.y * blockDim.y + threadIdx.y;

  CPtr<float> a = A[i];
  CPtr<float> b = &B[0][j];
  CPtr<float> c = &C[i][j];

  if (i<MAT_SIZE && j<MAT_SIZE) {
    float result = 0;
    for (size_t x=0; x<MAT_SIZE; ++x) {
      float fa = a.read();
      float fb = b.read();
      result += fa * fb;
      ++a;
      B+=MAT_SIZE;
    }
    c.write(result);
  }
}

extern "C" {

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    init
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_init
  (JNIEnv *, jclass)
{
  cpu_pointer::initialize();
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    clear
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_clear
  (JNIEnv *, jclass)
{
  cpu_pointer::clear_cache();
}


/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    fin
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_fin
  (JNIEnv *, jclass)
{
  cpu_pointer::finalize();
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    run
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_run
  (JNIEnv *, jclass)
{
  cpu_pointer::run_handler();
}

/*
 * Class:     org_sbsvm_Sbsvm
 * Method:    test
 * Signature: ()V
 */
JNIEXPORT void JNICALL Java_org_sbsvm_Sbsvm_test
  (JNIEnv *, jclass)
{
  float (*A)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  float (*B)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  float (*C)[MAT_SIZE] = new float[MAT_SIZE][MAT_SIZE];
  dim3 block(32, 8);
  dim3 grid(MAT_SIZE/block.x+(MAT_SIZE%block.x!=0), MAT_SIZE/block.y+(MAT_SIZE%block.y!=0));
  gpu_client<<<grid, block>>>(A, B, C);
}

} // extern "C"
